#include "hip/hip_runtime.h"
#include "CMassNormInc.h"
#include <memory.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace MassNorm;

static __global__ void mGSubtract(float* gfImg, int iPixels, float fVal)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= iPixels) return;
	else if(gfImg[i] < (float)-1e10) return;
	else gfImg[i] -= fVal;
}

GPositivity::GPositivity(void)
{
}

GPositivity::~GPositivity(void)
{
}

void GPositivity::DoIt(MrcUtil::CTomoStack* pTomoStack)
{
	printf("Set positivity ...\n");
	int* piStkSize = pTomoStack->m_aiStkSize;
	//---------------------------------------
	float *gfFrm = 0L;
	int iPixels = piStkSize[0] * piStkSize[1];
	size_t tBytes = sizeof(float) * iPixels;
	hipMalloc(&gfFrm, tBytes);
	//-------------------------
	bool bPadded = true, bSync = true;
	float fStackMin = (float)1e30;
	Util::GFindMinMax2D aFindMinMax;
	aFindMinMax.SetSize(piStkSize, !bPadded);
	//---------------------------------------
	for(int i=0; i<piStkSize[2]; i++)
	{	float* pfFrm = pTomoStack->GetFrame(i);
		hipMemcpy(gfFrm, pfFrm, tBytes, hipMemcpyDefault);
		//--------------------------------------------------
		float fMin = aFindMinMax.DoMin(gfFrm, bSync);
		float fMax = aFindMinMax.DoMax(gfFrm, bSync);
		if(fStackMin  > fMin) fStackMin = fMin;
		printf("%4d  %8.2f  %8.2f\n", i, fMin, fMax);
	}
	if(fStackMin >= 0)
	{	hipFree(gfFrm);
		printf("Positivity set.\n\n");
		return;
	}
	//-------------
	dim3 aBlockDim(512, 1);
	dim3 aGridDim(1, 1);
	aGridDim.x = (iPixels + aBlockDim.x - 1) / aBlockDim.x;
	for(int i=0; i<piStkSize[2]; i++)
	{	float* pfFrm = pTomoStack->GetFrame(i);
		hipMemcpy(gfFrm, pfFrm, tBytes, hipMemcpyDefault);
		mGSubtract<<<aGridDim, aBlockDim>>>(gfFrm, iPixels, fStackMin);
		hipMemcpy(pfFrm, gfFrm, tBytes, hipMemcpyDefault);
	}
	hipFree(gfFrm);
	printf("Positivity set.\n\n");
}

